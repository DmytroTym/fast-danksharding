#include "hip/hip_runtime.h"
#include "../../../icicle/icicle/curves/curve_config.cuh"
#include <hip/hip_runtime.h>

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

template <typename P>
void point_sum(P *h_outputs, P *h_inputs, unsigned nof_rows, unsigned nof_cols, unsigned l);

template<typename P>
__global__ void sum_kernel(P *d0, P *d1, unsigned nof_rows, unsigned nof_cols, unsigned l) {
  //Compute each thread's global row and column index
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned step = nof_cols / l;
  //Iterate over row, and down column
  d1[row * step + col] = P::zero();
  for (int k = 0; k < l; k++) {
    //Accumulate results for a single element
    d1[row * step + col] = d1[row * step + col] + d0[row * nof_cols + col + k * step];
  }
}

template <typename P>
void point_sum(P* h_outputs, P* h_inputs, unsigned nof_rows, unsigned nof_cols, unsigned l){
  unsigned step = nof_cols/l;
  P* d_outputs;
  P* d_inputs;
  hipMalloc(&d_outputs, sizeof(P) * nof_cols * step);
  hipMalloc(&d_inputs, sizeof(P) * nof_cols * nof_rows);
  //copy inputs to device
  hipMemcpy(d_inputs, h_inputs, sizeof(P) * nof_cols * nof_rows, hipMemcpyHostToDevice);

  unsigned NUM_THREADS = 1;
  unsigned THREADS_PER_ROW = (nof_rows + NUM_THREADS - 1) / NUM_THREADS;
  dim3 NUM_BLOCKS(THREADS_PER_ROW, step);

  sum_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(d_inputs, d_outputs, nof_rows, nof_cols, l);

  hipDeviceSynchronize();

  //check for errors
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
  }

  hipMemcpy(h_outputs, d_outputs, sizeof(P) * nof_rows * step, hipMemcpyDeviceToHost);

  //free memory
  hipFree(d_inputs);
  hipFree(d_outputs);
}

extern "C" int sum_of_points(projective_t *out, projective_t in[], size_t nof_rows, size_t nof_cols, size_t l, size_t device_id = 0)
{
    try
    {
        point_sum<projective_t>(out, in, nof_rows, nof_cols, l);

        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what()); // TODO: error code and message
        // out->z = 0; //TODO: .set_infinity()
    }
}

// the shared-memory version of matrix transpose taken from here: https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
template <typename T>
__global__ void transpose_kernel(T *odata, const T *idata)
{
  __shared__ T tile[TILE_DIM][TILE_DIM+1];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;
  int height = gridDim.y * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*height + x] = tile[threadIdx.x][threadIdx.y+j];
}

extern "C" int transpose_matrix(scalar_field_t *out, scalar_field_t *in, size_t nof_rows, size_t nof_cols, size_t device_id = 0)
{
    try
    {
        dim3 dimGrid(nof_rows / TILE_DIM, nof_cols / TILE_DIM, 1);
        dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
        transpose_kernel <scalar_t> <<<dimGrid, dimBlock>>> (out, in);

        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what()); // TODO: error code and message
    }
}
